#include "hip/hip_runtime.h"
/*
 * Example of how to use the mxGPUArray API in a MEX file.  This example shows
 * how to write a MEX function that takes a gpuArray input and returns a
 * gpuArray output, e.g. B=mexFunction(A).
 *
 * Copyright 2012 The MathWorks, Inc.
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdint.h>
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <cstdlib>
#include <algorithm>
#include <iostream>
using namespace std;

const int  Nthreads = 1024, maxFR = 500, NrankMax = 3;
//////////////////////////////////////////////////////////////////////////////////////////
__global__ void	Conv1D(const double *Params, const float *data, const float *W, float *conv_sig){    
  volatile __shared__ float  sW[81*NrankMax], sdata[(Nthreads+81)*NrankMax]; 
  float x, y;
  int tid, tid0, bid, i, nid, Nrank, NT, nt0;

  tid 		= threadIdx.x;
  bid 		= blockIdx.x;
  NT      	=   (int) Params[0];
  Nrank     = (int) Params[6];
  nt0       = (int) Params[4];
   
  if(tid<nt0*Nrank)
      sW[tid]= W[tid%nt0 + (tid/nt0) * nt0];
  __syncthreads();
  
  tid0 = 0;
  while (tid0<NT-Nthreads-nt0+1){
	  if (tid<nt0*NrankMax) 
          sdata[tid%nt0 + (tid/nt0)*(Nthreads+nt0)] = 
			data[tid0 + tid%nt0+ NT*bid];
	  
      #pragma unroll 3
      for(nid=0;nid<Nrank;nid++){
          sdata[tid + nt0+nid*(Nthreads+nt0)] = data[nt0+tid0 + tid+ NT*bid];
	  }
	  __syncthreads();
      
	  x = 0.0f;
      for(nid=0;nid<Nrank;nid++){
          y = 0.0f;
		  #pragma unroll 4
          for(i=0;i<nt0;i++)
              y    += sW[i + nid*nt0] * sdata[i+tid + nid*(Nthreads+nt0)];
          
          if (nid==0 && y<0)
              break;

           x += y*y;
      }
      conv_sig[tid0  + tid + NT*bid]   = x;
      
      tid0+=Nthreads;
      __syncthreads();
  }
}
//////////////////////////////////////////////////////////////////////////////////////////
__global__ void  bestFilter(const double *Params, const float *data, 
	float *err, int *ftype){
    
  int tid, tid0, i, bid, NT, Nchan, ibest = 0;
  float  Cf, Cbest = 0.0f;

  tid 		= threadIdx.x;
  bid 		= blockIdx.x;
  NT 		= (int) Params[0];
  Nchan     = (int) Params[9];
  
  tid0 = tid + bid * blockDim.x;
  while (tid0<NT){
      for (i=0; i<Nchan;i++){
          Cf = data[tid0 + NT * i];
          
          if (Cf > Cbest + 1e-6){
              Cbest 	= Cf;
              ibest 	= i;
          }
      }
      err[tid0] 	= Cbest;
      ftype[tid0] 	= ibest;
      
      tid0 += blockDim.x * gridDim.x;
  }
}

//////////////////////////////////////////////////////////////////////////////////////////
__global__ void	cleanup_spikes(const double *Params, const float *err, 
	const int *ftype, float *x, int *st, int *id, int *counter){
    
  int lockout, indx, tid, bid, NT, tid0,  j;
  volatile __shared__ float sdata[Nthreads+2*81+1];
  bool flag=0;
  float err0, Th;
  
  lockout   = (int) Params[4] - 1;
  tid 		= threadIdx.x;
  bid 		= blockIdx.x;
  
  NT      	=   (int) Params[0];
  tid0 		= bid * blockDim.x ;
  Th 		= (float) Params[2];
  
  while(tid0<NT-Nthreads-lockout+1){
      if (tid<2*lockout)
          sdata[tid] = err[tid0 + tid];
      sdata[tid+2*lockout] = err[2*lockout + tid0 + tid];
      
      __syncthreads();
      
      err0 = sdata[tid+lockout];
      if(err0>2*Th*Th){
          flag = 0;
          for(j=-lockout;j<=lockout;j++)
              if(sdata[tid+lockout+j]>err0){
                  flag = 1;
                  break;
              }
          if(flag==0){
              indx = atomicAdd(&counter[0], 1);
              if (indx<maxFR){
                  st[indx] = tid+lockout         + tid0;
                  id[indx] = ftype[tid+lockout   + tid0];
                  x[indx]  = err0;
              }
          }
      }
      
      tid0 += blockDim.x * gridDim.x;
  }
}

//////////////////////////////////////////////////////////////////////////////////////////
__global__ void	cleanup_heights(const double *Params, const float *x, 
        const int *st, const int *id, int *st1, int *id1, int *counter){
    
  int indx, tid, bid, t, d;
  volatile __shared__ float s_id[maxFR], s_x[maxFR];
  bool flag=0;
  float xmax;
  
  tid 		= threadIdx.x;
  bid 		= blockIdx.x;
    
  while (tid<counter[0]){
      s_x[tid]  = x[tid];
      s_id[tid] = id[tid];
      tid+=blockDim.x;
  }
  __syncthreads();
   
  tid = bid*blockDim.x + threadIdx.x;
          
  if (tid<counter[0]){
      xmax = s_x[tid];
      flag = 1;
      for (t=0; t<counter[0];t++){
          d = abs(s_id[t] - s_id[tid]);
          if (d<5 && xmax< s_x[t]){
              flag = 0;
                break;
          }   
      }
      // if flag, then your thread is the max across nearby channels
      if(flag){
          indx = atomicAdd(&counter[1], 1);
          st1[indx] = st[tid];
          id1[indx] = s_id[tid];
      }
  }
  
}


//////////////////////////////////////////////////////////////////////////////////////////
__global__ void extract_snips(const double *Params, const int *st, const int *id,
        const int *counter, const float *dataraw,  float *WU){
    
  int nt0, tidx, tidy, bid, ind, NT, Nchan;
  
  NT        = (int) Params[0];
  nt0       = (int) Params[4];
  Nchan     = (int) Params[9];
   
  tidx 		= threadIdx.x;
  bid 		= blockIdx.x;
  
  for(ind=0; ind<counter[1];ind++)
      if (id[ind]==bid){
		  tidy 		= threadIdx.y;
		  while (tidy<Nchan){	
            WU[tidx+tidy*nt0 + nt0*Nchan * ind] = dataraw[st[ind]+tidx + NT * tidy];
			tidy+=blockDim.y;
		  }
	  }
}
//////////////////////////////////////////////////////////////////////////////////////////

/*
 * Host code
 */
void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
  /* Initialize the MathWorks GPU API. */
  mxInitGPU();

  /* Declare input variables*/
  double *Params, *d_Params;
  int nt0, NT, Nchan, Nnearest;

  
  /* read Params and copy to GPU */
  Params  	= (double*) mxGetData(prhs[0]);
  NT		= (int) Params[0];
  Nchan     = (int) Params[9];
  nt0       = (int) Params[4];
  Nnearest  = (int) Params[5];
  
  dim3 tpB(8, 2*nt0-1), tpF(16, Nnearest), tpS(nt0, 16);
        
  hipMalloc(&d_Params,      sizeof(double)*mxGetNumberOfElements(prhs[0]));
  hipMemcpy(d_Params,Params,sizeof(double)*mxGetNumberOfElements(prhs[0]),hipMemcpyHostToDevice);

   /* collect input GPU variables*/
  mxGPUArray const  *W,  *data;
  const float     *d_W, *d_data;
  
  data       = mxGPUCreateFromMxArray(prhs[1]);
  d_data     = (float const *)(mxGPUGetDataReadOnly(data));
  W             = mxGPUCreateFromMxArray(prhs[2]);
  d_W        	= (float const *)(mxGPUGetDataReadOnly(W));
  
  
  /* allocate new GPU variables*/  
  float *d_err,*d_x, *d_dout, *d_WU;
  int *d_st, *d_id1, *d_st1, *d_ftype,  *d_id, *d_counter;

  hipMalloc(&d_dout,   NT * Nchan* sizeof(float));
  hipMalloc(&d_err,   NT * sizeof(float));
  hipMalloc(&d_ftype, NT * sizeof(int));  
  hipMalloc(&d_st,    maxFR * sizeof(int));
  hipMalloc(&d_id,    maxFR * sizeof(int));
  hipMalloc(&d_st1,    maxFR * sizeof(int));
  hipMalloc(&d_id1,    maxFR * sizeof(int));
  hipMalloc(&d_x,     maxFR * sizeof(float));
  
  hipMalloc(&d_WU,    maxFR*nt0*Nchan * sizeof(float));
  hipMalloc(&d_counter,   2*sizeof(int));
  
  hipMemset(d_WU,      0, maxFR*nt0*Nchan * sizeof(float));
  hipMemset(d_counter, 0, 2*sizeof(int));
  hipMemset(d_dout,    0, NT * Nchan * sizeof(float));
  hipMemset(d_err,     0, NT * sizeof(float));
  hipMemset(d_ftype,   0, NT * sizeof(int));
  hipMemset(d_st,      0, maxFR *   sizeof(int));
  hipMemset(d_id,      0, maxFR *   sizeof(int));
  hipMemset(d_st1,      0, maxFR *   sizeof(int));
  hipMemset(d_id1,      0, maxFR *   sizeof(int));
  hipMemset(d_x,      0, maxFR *   sizeof(float));
  
  
  int *counter;
  counter = (int*) calloc(1,sizeof(int));
  
  // filter the data with the temporal templates
  Conv1D<<<Nchan, Nthreads>>>(d_Params, d_data, d_W, d_dout);
  
  // compute the best filter
  bestFilter<<<NT/Nthreads,Nthreads>>>(d_Params, d_dout, d_err, d_ftype);
  
  // ignore peaks that are smaller than another nearby peak
  cleanup_spikes<<<NT/Nthreads,Nthreads>>>(d_Params,
          d_err, d_ftype, d_x, d_st, d_id, d_counter);
  
  // ignore peaks that are smaller than another nearby peak
  cleanup_heights<<<1 + maxFR/32, 32>>>(d_Params, d_x, d_st, d_id, d_st1, d_id1, d_counter);
  
  // add new spikes to 2nd counter
  hipMemcpy(counter,     d_counter+1, sizeof(int), hipMemcpyDeviceToHost);
  
  // update dWU here by adding back to subbed spikes
  extract_snips<<<Nchan,tpS>>>(  d_Params, d_st1, d_id1, d_counter, d_data, d_WU);
  
  mxGPUArray *WU1;
  float  *d_WU1;
  const mwSize dimsu[] 	= {nt0, Nchan, counter[0]};
  WU1 		= mxGPUCreateGPUArray(3, dimsu, mxSINGLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);  
  d_WU1 		= (float *)(mxGPUGetData(WU1));
  
  hipMemcpy(d_WU1, d_WU, nt0*Nchan*counter[0]*sizeof(float), hipMemcpyDeviceToDevice);
  
  // dWU stays a GPU array
  plhs[0] 	= mxGPUCreateMxArrayOnGPU(WU1);

  
  hipFree(d_ftype);
  hipFree(d_err);
  hipFree(d_st);
  hipFree(d_id);
  hipFree(d_st1);
  hipFree(d_x);
  hipFree(d_id1);
  hipFree(d_counter);
  hipFree(d_Params);
  
  hipFree(d_dout);
  hipFree(d_WU);
  
  mxGPUDestroyGPUArray(W);
  mxGPUDestroyGPUArray(data);
  mxGPUDestroyGPUArray(WU1);
}